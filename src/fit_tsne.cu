#include "hip/hip_runtime.h"
/*
    Compute t-SNE via Barnes-Hut for NlogN time.
*/

#include "include/fit_tsne.h"
#include <chrono>
#include <string>

#define START_IL_REORDER() startReorder = std::chrono::high_resolution_clock::now();
#define END_IL_REORDER(x) endReorder = std::chrono::high_resolution_clock::now(); duration = std::chrono::duration_cast<std::chrono::microseconds>(endReorder-startReorder); x += duration; total_time += duration;
#define START_IL_TIMER() start = std::chrono::high_resolution_clock::now();
#define END_IL_TIMER(x) stop = std::chrono::high_resolution_clock::now(); duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start); x += duration; total_time += duration;
#define PRINT_IL_TIMER(x) std::cout << #x << ": " << ((float) x.count()) / 1000000.0 << "s" << std::endl

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed with error (%d) at line %d\n",             \
               status, __LINE__);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

//Custom comparator for permuting a thrust vector
struct copy_idx_func : public thrust::unary_function<unsigned, unsigned>
{
  size_t c;
  unsigned *p;
  copy_idx_func(const size_t _c, unsigned *_p) : c(_c), p(_p) {};
  __host__ __device__
    unsigned operator()(unsigned idx) {
      unsigned myrow = idx/c;
      unsigned newrow = p[myrow] - 1;
      unsigned mycol = idx%c;
      return newrow*c+mycol;
    }
};
//Save GPU array to file
template <typename T>
void tsnecuda::save_coo(std::string filename, thrust::device_vector<T> device_vec, int size_coo ) {
    std::ofstream dump_coo;
    T *h_coo = (T *)malloc((size_coo * 2)*sizeof(T));
    hipMemcpy(h_coo, thrust::raw_pointer_cast(device_vec.data()), sizeof(T)*(size_coo*2), hipMemcpyDeviceToHost);

    dump_coo.open(filename + std::to_string(size_coo));
    for(int i=0;i<size_coo*2;i++) {
      dump_coo << h_coo[i] << " ";
    }
    dump_coo.close();
}

//Split string
std::vector<std::string> tsnecuda::split (std::string s, std::string delimiter) {
    size_t pos_start = 0, pos_end, delim_len = delimiter.length();
    std::string token;
    std::vector<std::string> res;

    while ((pos_end = s.find (delimiter, pos_start)) != std::string::npos) {
        token = s.substr (pos_start, pos_end - pos_start);
        pos_start = pos_end + delim_len;
        res.push_back (token);
    }

    res.push_back (s.substr (pos_start));
    return res;
}
void tsnecuda::RunTsne(tsnecuda::Options &opt,
                       tsnecuda::GpuOptions &gpu_opt)
{
    auto start = std::chrono::high_resolution_clock::now();
    auto stop = std::chrono::high_resolution_clock::now();
    auto endReorder = std::chrono::high_resolution_clock::now();
    auto startReorder = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    
    auto total_time = duration;
    auto _time_initialization = duration;
    auto _time_knn = duration;
    auto _time_knn2 = duration;
    auto _time_normknn = duration;
    auto _time_symmetry = duration;
    auto _time_perm = duration;
    auto _time_reorder = duration;
    auto _time_reord_buff = duration;
    auto _time_mapping = duration;
    auto _time_devicecopy = duration;
    auto _time_hostcopy = duration;
    auto _time_tot_perm = duration;
    auto _time_init_low_dim = duration;
    auto _time_init_fft = duration;
    auto _time_precompute_2d = duration;
    auto _time_nbodyfft = duration;
    auto _time_compute_charges = duration;
    auto _time_other = duration;
    auto _time_norm = duration;
    auto _time_attr = duration;
    auto _time_apply_forces = duration;

    // Check the validity of the options file
    if (!opt.validate()) {
        std::cout << "E: Invalid options file. Terminating." << std::endl;
        return;
    }

    START_IL_TIMER();

    if (opt.verbosity > 0) {
        std::cout << "Initializing cuda handles... " << std::flush;
    }

    // Construct the handles
    hipblasHandle_t dense_handle;
    CublasSafeCall(hipblasCreate(&dense_handle));
    hipsparseHandle_t sparse_handle;
    CusparseSafeCall(hipsparseCreate(&sparse_handle));
    std::cout << "Created cublas handle" << std::endl;
    // Set CUDA device properties
    const int num_blocks = gpu_opt.sm_count;

    // Construct sparse matrix descriptor
    hipsparseMatDescr_t sparse_matrix_descriptor;

    //hipsparseDnMatDescr_t dense_pts;
    //hipsparseDnMatDescr_t dense_pijqij;

    hipsparseCreateMatDescr(&sparse_matrix_descriptor);
        
    
    hipsparseSetMatType(sparse_matrix_descriptor, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(sparse_matrix_descriptor,HIPSPARSE_INDEX_BASE_ZERO);

    // Setup some return information if we're working on snapshots
    int snap_num = 0;
    int snap_interval = 1;
    if (opt.return_style == tsnecuda::RETURN_STYLE::SNAPSHOT) {
        snap_interval = opt.iterations / (opt.num_snapshots - 1);
    }

    // Get constants from options
    const int num_points = opt.num_points;
    const int num_neighbors = (opt.num_neighbors < num_points) ? opt.num_neighbors : num_points;
    const float *high_dim_points = opt.points;
    const int high_dim = opt.num_dims;
    const float perplexity = opt.perplexity;
    const float perplexity_search_epsilon = opt.perplexity_search_epsilon;
    const float eta = opt.learning_rate;
    float momentum = opt.pre_exaggeration_momentum;
    float attr_exaggeration = opt.early_exaggeration;
    float normalization;

    // Allocate host memory
    float *knn_squared_distances = new float[num_points * num_neighbors];
    memset(knn_squared_distances, 0, num_points * num_neighbors * sizeof(float));
    long *knn_indices = new long[num_points * num_neighbors];

    // Set cache configs
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::IntegrationKernel), hipFuncCachePreferL1);
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::ComputePijxQijKernel), hipFuncCachePreferShared);
    GpuErrorCheck(hipDeviceSynchronize());


    END_IL_TIMER(_time_initialization);
    START_IL_TIMER();

    if (opt.verbosity > 0) {
        std::cout << "done.\nKNN Computation... " << std::flush;
    }
    // Compute approximate K Nearest Neighbors and squared distances
    tsnecuda::util::KNearestNeighbors(gpu_opt, knn_indices, knn_squared_distances, high_dim_points, high_dim, num_points, num_neighbors);
    END_IL_TIMER(_time_knn);
    START_IL_TIMER();
    thrust::device_vector<long> knn_indices_long_device(knn_indices, knn_indices + num_points * num_neighbors);
    thrust::device_vector<int> knn_indices_device(num_points * num_neighbors);
    tsnecuda::util::PostprocessNeighborIndices(gpu_opt, knn_indices_device, knn_indices_long_device,
                                                        num_points, num_neighbors);
    END_IL_TIMER(_time_knn2);
    START_IL_TIMER();
    // Max-norm the distances to avoid exponentiating by large numbers
    thrust::device_vector<float> knn_squared_distances_device(knn_squared_distances,
                                            knn_squared_distances + (num_points * num_neighbors));
    tsnecuda::util::MaxNormalizeDeviceVector(knn_squared_distances_device);

    END_IL_TIMER(_time_normknn);
    START_IL_TIMER();

    if (opt.verbosity > 0) {
        std::cout << "done.\nComputing Pij matrix... " << std::flush;
    }

    // Search Perplexity
    thrust::device_vector<float> pij_non_symmetric_device(num_points * num_neighbors);
    tsnecuda::SearchPerplexity(gpu_opt, dense_handle, pij_non_symmetric_device, knn_squared_distances_device,
                                    perplexity, perplexity_search_epsilon, num_points, num_neighbors);

    // Clean up memory
    knn_squared_distances_device.clear();
    knn_squared_distances_device.shrink_to_fit();
    knn_indices_long_device.clear();
    knn_indices_long_device.shrink_to_fit();
    delete[] knn_squared_distances;

    if(opt.reorder == 7){
      //Dump knn info to file
      std::ofstream knn_file;
      knn_file.open("knn_" + std::to_string(opt.num_points));
      //host_ys = new float[num_points * 2];
      //dump_file << num_points << " " << 2 << std::endl;
      for(int i=0; i<num_points*num_neighbors; i++){
        knn_file << knn_indices[i] << " ";
      }

    }
    delete[] knn_indices;

    // Symmetrize the pij matrix
    thrust::device_vector<float> sparse_pij_device;
    thrust::device_vector<int> pij_row_ptr_device;
    thrust::device_vector<int> pij_col_ind_device;
    tsnecuda::util::SymmetrizeMatrix(sparse_handle, sparse_pij_device, pij_row_ptr_device,
                                        pij_col_ind_device, pij_non_symmetric_device, knn_indices_device,
                                        opt.magnitude_factor, num_points, num_neighbors);

    const int num_nonzero = sparse_pij_device.size();
        // Clean up memory
    knn_indices_device.clear();
    knn_indices_device.shrink_to_fit();
    pij_non_symmetric_device.clear();
    pij_non_symmetric_device.shrink_to_fit();

    // Declare memory

    //thrust::device_vector<float> pijqij(sparse_pij_device.size());
    thrust::device_vector<float> repulsive_forces_device(opt.num_points * 2, 0);
    thrust::device_vector<float> attractive_forces_device(opt.num_points * 2, 0);
    thrust::device_vector<float> gains_device(opt.num_points * 2, 1);
    thrust::device_vector<float> old_forces_device(opt.num_points * 2, 0); // for momentum
    thrust::device_vector<float> normalization_vec_device(opt.num_points);
    thrust::device_vector<float> ones_device(opt.num_points * 2, 1); // This is for reduce summing, etc.
    thrust::device_vector<int> coo_indices_device(sparse_pij_device.size()*2);

    //tsnecuda::util::Csr2Coo(gpu_opt, coo_indices_device, pij_row_ptr_device,
      //                      pij_col_ind_device, num_points, num_nonzero);

    END_IL_TIMER(_time_symmetry);
    START_IL_TIMER();
    
        
    
    

    if (opt.verbosity > 0) {
        std::cout << "done.\nInitializing low dim points... " << std::flush;
        //std::ifstream re_pij;
        //re_pij.open("./re_pij.txt");

    }

    // Initialize Low-Dim Points
    thrust::device_vector<float> points_device(num_points * 2);
    thrust::device_vector<float> random_vector_device(points_device.size());

    std::default_random_engine generator(opt.random_seed);
    std::normal_distribution<float> distribution1(0.0, 1.0);
    thrust::host_vector<float> h_points_device(num_points * 2);

    
    // Initialize random noise vector
    for (int i = 0; i < h_points_device.size(); i++) h_points_device[i] = 0.001 * distribution1(generator);
    thrust::copy(h_points_device.begin(), h_points_device.end(), random_vector_device.begin());

    // TODO: this will only work with gaussian init
    if (opt.initialization == tsnecuda::TSNE_INIT::UNIFORM) { // Random uniform initialization
        points_device = tsnecuda::util::RandomDeviceVectorInRange(generator, points_device.size(), -5, 5);
    } else if (opt.initialization == tsnecuda::TSNE_INIT::GAUSSIAN) { // Random gaussian initialization
        // Generate some Gaussian noise for the points
        for (int i = 0; i < h_points_device.size(); i++) h_points_device[i] = 0.0001 * distribution1(generator);
        thrust::copy(h_points_device.begin(), h_points_device.end(), points_device.begin());
    } else if (opt.initialization == tsnecuda::TSNE_INIT::RESUME) { // Preinit from vector
        // Load from vector
        if(opt.preinit_data != nullptr) {
          thrust::copy(opt.preinit_data, opt.preinit_data + points_device.size(), points_device.begin());
        } else {
          std::cerr << "E: Invalid initialization. Initialization points are null." << std::endl;
          exit(1);
        }
    } else if (opt.initialization == tsnecuda::TSNE_INIT::VECTOR) { // Preinit from vector points only
        // Copy the pre-init data
        if(opt.preinit_data != nullptr) {
          thrust::copy(opt.preinit_data, opt.preinit_data + points_device.size(), points_device.begin());
        } else {
          std::cerr << "E: Invalid initialization. Initialization points are null." << std::endl;
          exit(1);
        }
    } else { // Invalid initialization
        std::cerr << "E: Invalid initialization type specified." << std::endl;
        exit(1);
    }

    END_IL_TIMER(_time_init_low_dim);
    START_IL_TIMER();

    if (opt.verbosity > 0) {
        std::cout << "done.\nInitializing CUDA memory... " << std::flush;
    }
    
    //Define the cuSparse matrices
    //
    //Create the cuSparse matrix (CSR)
    //hipsparseCreateMatDescr(&sparse_matrix_descriptor);
    //hipsparseSetMatType(sparse_matrix_descriptor, HIPSPARSE_MATRIX_TYPE_GENERAL);
    //hipsparseSetMatIndexBase(sparse_matrix_descriptor,
      //  HIPSPARSE_INDEX_BASE_ZERO);
    
    
      float *h_pij_vals2 = (float *)malloc((num_nonzero)*sizeof(float));
      //h_pij_vals = thrust::raw_pointer_cast(sparse_pij_device.data());
      hipMemcpy(h_pij_vals2, thrust::raw_pointer_cast(sparse_pij_device.data()), sizeof(float)*(num_nonzero), hipMemcpyDeviceToHost);
     
      int *h_pij_row_ptr2 = (int *)malloc((num_points+1)*sizeof(int));
      //h_pij_row_ptr = thrust::raw_pointer_cast(pij_row_ptr_device.data());
      hipMemcpy(h_pij_row_ptr2, thrust::raw_pointer_cast(pij_row_ptr_device.data()), sizeof(int)*(num_points+1), hipMemcpyDeviceToHost);

      int *h_pij_col_ind2 = (int *)malloc((num_nonzero)*sizeof(int));
      //h_pij_col_ind = thrust::raw_pointer_cast(pij_col_ind_device.data());
      hipMemcpy(h_pij_col_ind2, thrust::raw_pointer_cast(pij_col_ind_device.data()), sizeof(int)*(num_nonzero), hipMemcpyDeviceToHost);

    std::ofstream vals_file;
    std::ofstream row_file;
    std::ofstream ind_file;
    vals_file.open("vals_" + std::to_string(opt.num_points));
    row_file.open("rows_" + std::to_string(opt.num_points));
    ind_file.open("ind_" + std::to_string(opt.num_points));

      for(int i=0; i<num_nonzero; i++){
        vals_file << h_pij_vals2[i] << " ";
      }
      for(int i=0; i<num_points+1; i++){
        row_file << h_pij_row_ptr2[i] << " ";
      }
      for(int i=0; i<num_nonzero; i++){
        ind_file << h_pij_col_ind2[i] << " ";
      }


    //permute the pij sparse matrix
    std::cout << "Num num_nonzero: " << num_nonzero << std::endl;
    tsnecuda::util::Csr2Coo(gpu_opt, coo_indices_device, pij_row_ptr_device, pij_col_ind_device, num_points, num_nonzero);
    
    tsnecuda::save_coo("coo_before_", coo_indices_device, num_nonzero);
    START_IL_REORDER();
    if(opt.reorder==1) {
      START_IL_TIMER();
      int issym = 0;
      int *h_Q = NULL;
      //int *h_pij_row_ptr_b = NULL;
      int *h_mapBfromA = NULL;
      //float *h_pij_vals_b = NULL;
      //int *h_pij_col_ind_b = NULL;      
      hipsolverSpHandle_t sol_handle = NULL;
      checkCudaErrors(hipsolverSpCreate(&sol_handle));
       std::cout << "Created sparse solver handle" << std::endl;
      float *h_pij_vals = (float *)malloc((num_nonzero)*sizeof(float));
      //h_pij_vals = thrust::raw_pointer_cast(sparse_pij_device.data());
      hipMemcpy(h_pij_vals, thrust::raw_pointer_cast(sparse_pij_device.data()), sizeof(float)*(num_nonzero), hipMemcpyDeviceToHost);
     
      int *h_pij_row_ptr = (int *)malloc((num_points+1)*sizeof(int));
      //h_pij_row_ptr = thrust::raw_pointer_cast(pij_row_ptr_device.data());
      hipMemcpy(h_pij_row_ptr, thrust::raw_pointer_cast(pij_row_ptr_device.data()), sizeof(int)*(num_points+1), hipMemcpyDeviceToHost);

      int *h_pij_col_ind = (int *)malloc((num_nonzero)*sizeof(int));
      //h_pij_col_ind = thrust::raw_pointer_cast(pij_col_ind_device.data());
      hipMemcpy(h_pij_col_ind, thrust::raw_pointer_cast(pij_col_ind_device.data()), sizeof(int)*(num_nonzero), hipMemcpyDeviceToHost);

      h_Q = (int *)malloc(sizeof(int)*num_points);
      //h_pij_row_ptr_b = (int *)malloc(sizeof(int)*(num_points+1));
      //h_pij_col_ind_b = (int *)malloc(sizeof(int)*(num_nonzero));
      //h_pij_vals_b = (float *)malloc(sizeof(float)*(num_nonzero));
      h_mapBfromA = (int *)malloc(sizeof(int)*num_nonzero);
      
      //check if memory has been allocated without any issues
      assert(NULL != h_Q);
      //assert(NULL != h_pij_row_ptr_b);
      //assert(NULL != h_pij_col_ind_b);
      //assert(NULL != h_pij_vals_b   );
      assert(NULL != h_mapBfromA);
      
      std::cout << "Assertion done" << std::endl;
      END_IL_TIMER(_time_hostcopy);      
      //Compute the permutation vector
      std::cout << "Permuting matrix...";
      START_IL_TIMER();
      if(opt.reopt == 0) {                // RCM
        checkCudaErrors(cusolverSpXcsrsymrcmHost(sol_handle, num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, h_Q));

      }
      else{
        checkCudaErrors(cusolverSpXcsrsymamdHost(sol_handle, num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, h_Q));
        
      }
      END_IL_TIMER(_time_perm);
      std::cout << "Permutation computed..." << std::endl;
      
      //float *h_pts_perm = (float *)malloc(sizeof(float)*(num_points*2));
      //checkCudaErrors(hipMemcpy(h_pts_perm, thrust::raw_pointer_cast(points_device.data()), sizeof(float)*(num_points*2), hipMemcpyDeviceToHost));
      //float *h_pts = (float *)malloc(sizeof(float)*(num_points*2));
      //checkCudaErrors(hipMemcpy(h_pts, thrust::raw_pointer_cast(points.device.data()), sizeof(float)*(num_points*2), hipMemcpyDeviceToHost));

      //for (int i=0; i < num_points*2; i += 2){
      	
      //}
      //memcpy(h_pij_row_ptr_b, h_pij_row_ptr, sizeof(int)*(num_points+1));
      //memcpy(h_pij_col_ind_b, h_pij_col_ind, sizeof(int)*num_nonzero);
      
      size_t size_perm = 0;
      void *buffer_cpu = NULL;
      START_IL_TIMER();
      checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(sol_handle, num_points, num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, h_Q, h_Q, &size_perm));
      END_IL_TIMER(_time_reord_buff);
      buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
      assert(NULL!=buffer_cpu);


      for(int j = 0 ; j < num_nonzero ; j++)
      {
        h_mapBfromA[j] = j;
      }
      START_IL_TIMER();
      checkCudaErrors(cusolverSpXcsrpermHost(sol_handle, num_points, num_points, num_nonzero ,sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, h_Q, h_Q, h_mapBfromA, buffer_cpu));
      END_IL_TIMER(_time_reorder);
      //Map the values
      START_IL_TIMER();
      for(int j = 0 ; j < num_nonzero ; j++)
      {
            h_pij_vals[j] = h_pij_vals[ h_mapBfromA[j] ];
      }
	    END_IL_TIMER(_time_mapping);
      //memcpy(h_pij_row_ptr, h_pij_row_ptr_b, sizeof(int)*(num_points+1));
      //memcpy(h_pij_col_ind, h_pij_col_ind_b, sizeof(int)*num_nonzero);
      //memcpy(h_pij_vals, h_pij_vals_b, sizeof(float)*num_nonzero);

      //delete [] h_pij_row_ptr_b;
      //delete [] h_pij_col_ind_b;
      //delete [] h_pij_vals_b;
      //std::ofstream reord_file;
      //reord_file.open("reordering_rcm.txt");
      //for (int i=0; i < num_points; i++) {
        //reord_file << h_Q[i] << " ";
      //}

      delete [] h_mapBfromA;
      delete [] h_Q;
      if (buffer_cpu) {free(buffer_cpu);}
      if (sol_handle) { checkCudaErrors(hipsolverSpDestroy(sol_handle)); }


      
      
      std::cout << "Matrix B created" << std::endl;
      START_IL_TIMER();
      //int *d_pij_row_ptr;
      //checkCudaErrors(hipMalloc((void**)&d_pij_row_ptr, sizeof(int)*(num_points+1)));
      //checkCudaErrors(hipMemcpy(d_pij_row_ptr, h_pij_row_ptr, sizeof(int)*(num_points+1) ,hipMemcpyHostToDevice));
      //thrust::device_ptr<int> dp_row = thrust::device_pointer_cast(d_pij_row_ptr);
      std::vector<int> v_row_ptr(h_pij_row_ptr, h_pij_row_ptr + (num_points+1));
      if (h_pij_row_ptr) { free(h_pij_row_ptr); }
      thrust::host_vector<int> row_temp(v_row_ptr);

      //int *d_pij_col_ind;
      //checkCudaErrors(hipMalloc((void**)&d_pij_col_ind, sizeof(int)*(num_nonzero)));
      //checkCudaErrors(hipMemcpy(d_pij_col_ind, h_pij_col_ind, sizeof(int)*(num_nonzero), hipMemcpyHostToDevice));
      //thrust::device_ptr<int> dp_col = thrust::device_pointer_cast(d_pij_col_ind);
      std::vector<int> v_col_ind(h_pij_col_ind, h_pij_col_ind + (num_nonzero));
      if (h_pij_col_ind) { free(h_pij_col_ind); }
      thrust::host_vector<int> col_temp(v_col_ind);
      
      //float *d_pij_vals;
      //checkCudaErrors(hipMalloc((void**)&d_pij_vals, sizeof(float)*(num_nonzero)));
      //checkCudaErrors(hipMemcpy(d_pij_vals, h_pij_vals,sizeof(float)*(num_nonzero), hipMemcpyHostToDevice));
      //thrust::device_ptr<float> dp_vals(d_pij_vals);
      std::vector<float> v_vals(h_pij_vals, h_pij_vals + (num_nonzero+1));
      if (h_pij_vals) {free(h_pij_vals);}
      thrust::host_vector<float> vals_temp(v_vals);
      //Update Pij vector to be passed to ComputeAttractiveForces
      
      
      pij_row_ptr_device = row_temp;
      pij_col_ind_device = col_temp;
      sparse_pij_device = vals_temp;
      //thru
      //st::copy(col_temp.begin(), col_temp.end() , pij_col_ind_device.begin());
      //thrust::copy(vals_temp.begin(), vals_temp.end(), sparse_pij_device.begin());
      END_IL_TIMER(_time_devicecopy);
      std::cout << "Completed permuting" << std::endl;
      // Free memory
                       //if (h_pij_vals) {free(h_pij_vals); }
      
      //if (h_pij_row_ptr_b) { free(h_pij_row_ptr_b); }
      //if (h_pij_col_ind_b) { free(h_pij_col_ind_b); }
      //if (h_pij_vals_b) {free(h_pij_vals_b); }

      //if (h_Q) { free(h_Q); }
      //if (buffer_cpu) {free(buffer_cpu);}

      //if (d_pij_row_ptr) { checkCudaErrors(hipFree(d_pij_row_ptr));}
      //if (d_pij_col_ind) { checkCudaErrors(hipFree(d_pij_col_ind));} 
      //if (d_pij_vals) { checkCudaErrors(hipFree(d_pij_vals));} 

    }
    else if (opt.reorder==2){
      //hipsolverStatus_t status_nd = NULL;
      
      hipsolverSpHandle_t sol_handle = NULL;
      checkCudaErrors(hipsolverSpCreate(&sol_handle));
      
      
      int *h_pij_row_ptr = (int *)malloc((num_points+1)*sizeof(int));
      //h_pij_row_ptr = thrust::raw_pointer_cast(pij_row_ptr_device.data());
      hipMemcpy(h_pij_row_ptr, thrust::raw_pointer_cast(pij_row_ptr_device.data()), sizeof(int)*(num_points+1), hipMemcpyDeviceToHost);

      int *h_pij_col_ind = (int *)malloc((num_nonzero)*sizeof(int));
      //h_pij_col_ind = thrust::raw_pointer_cast(pij_col_ind_device.data());
      hipMemcpy(h_pij_col_ind, thrust::raw_pointer_cast(pij_col_ind_device.data()), sizeof(int)*(num_nonzero), hipMemcpyDeviceToHost);
      
      float *h_pij_vals = (float*)malloc((num_nonzero)*sizeof(float));
      hipMemcpy(h_pij_vals, thrust::raw_pointer_cast(sparse_pij_device.data()), sizeof(float)*(num_nonzero), hipMemcpyDeviceToHost);

      int *h_Q = (int *)malloc(sizeof(int)*num_points);
      int *h_mapBfromA = NULL;

      h_mapBfromA = (int *)malloc(sizeof(int)*num_nonzero);

      
      
      cusolverSpXcsrmetisndHost(sol_handle, num_points,num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, NULL, h_Q);
      
      size_t size_perm = 0;
      void *buffer_cpu = NULL;
      START_IL_TIMER();
      checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(sol_handle, num_points, num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, h_Q, h_Q, &size_perm));
      END_IL_TIMER(_time_reord_buff);
      buffer_cpu = (void*)malloc(sizeof(char)*size_perm);


      START_IL_TIMER();
      checkCudaErrors(cusolverSpXcsrpermHost(sol_handle, num_points,num_points, num_nonzero ,sparse_matrix_descriptor, h_pij_row_ptr,h_pij_col_ind, h_Q, h_Q, NULL, buffer_cpu));
      END_IL_TIMER(_time_reorder);
      
       //Map the values
      START_IL_TIMER();
      for(int j = 0 ; j < num_nonzero ; j++)
      {
            h_pij_vals[j] = h_pij_vals[ h_mapBfromA[j] ];
      }
	    END_IL_TIMER(_time_mapping);


      delete [] h_mapBfromA;
      delete [] h_Q;
      if (buffer_cpu) {free(buffer_cpu);}
      if (sol_handle) { checkCudaErrors(hipsolverSpDestroy(sol_handle)); }

      START_IL_TIMER();
      //int *d_pij_row_ptr;
      //checkCudaErrors(hipMalloc((void**)&d_pij_row_ptr, sizeof(int)*(num_points+1)));
      //checkCudaErrors(hipMemcpy(d_pij_row_ptr, h_pij_row_ptr, sizeof(int)*(num_points+1) ,hipMemcpyHostToDevice));
      //thrust::device_ptr<int> dp_row = thrust::device_pointer_cast(d_pij_row_ptr);
      std::vector<int> v_row_ptr(h_pij_row_ptr, h_pij_row_ptr + (num_points+1));
      if (h_pij_row_ptr) { free(h_pij_row_ptr); }
      thrust::host_vector<int> row_temp(v_row_ptr);

      //int *d_pij_col_ind;
      //checkCudaErrors(hipMalloc((void**)&d_pij_col_ind, sizeof(int)*(num_nonzero)));
      //checkCudaErrors(hipMemcpy(d_pij_col_ind, h_pij_col_ind, sizeof(int)*(num_nonzero), hipMemcpyHostToDevice));
      //thrust::device_ptr<int> dp_col = thrust::device_pointer_cast(d_pij_col_ind);
      std::vector<int> v_col_ind(h_pij_col_ind, h_pij_col_ind + (num_nonzero));
      if (h_pij_col_ind) { free(h_pij_col_ind); }
      thrust::host_vector<int> col_temp(v_col_ind);
      
      //float *d_pij_vals;
      //checkCudaErrors(hipMalloc((void**)&d_pij_vals, sizeof(float)*(num_nonzero)));
      //checkCudaErrors(hipMemcpy(d_pij_vals, h_pij_vals,sizeof(float)*(num_nonzero), hipMemcpyHostToDevice));
      //thrust::device_ptr<float> dp_vals(d_pij_vals);
      std::vector<float> v_vals(h_pij_vals, h_pij_vals + (num_nonzero+1));
      if (h_pij_vals) {free(h_pij_vals);}
      thrust::host_vector<float> vals_temp(v_vals);
      //Update Pij vector to be passed to ComputeAttractiveForces
      
      
      pij_row_ptr_device = row_temp;
      pij_col_ind_device = col_temp;
      sparse_pij_device = vals_temp;
      //thru
      //st::copy(col_temp.begin(), col_temp.end() , pij_col_ind_device.begin());
      //thrust::copy(vals_temp.begin(), vals_temp.end(), sparse_pij_device.begin());
      END_IL_TIMER(_time_devicecopy);
      std::cout << "Completed permuting" << std::endl;


    }
    //coloring
    
    else if(opt.reorder == 3) {
      int ncolors=0; //coloring[num_points] = {0}, reordering[num_points] = {0};
      
      int *coloring = (int *)malloc(num_points * sizeof(int));
      int *reordering = (int *)malloc(num_points * sizeof(int));

      float fraction=1.0;
      int *d_coloring, *d_reordering;
      //float *d_fraction;
      //int *d_num_points, *d_nnz;
      
      //hipMalloc((void **)&d_num_points, sizeof(int));
      //hipMalloc((void **)&d_nnz, sizeof(int));

      //hipMalloc((void **)&d_ncolors, sizeof(int));
      hipMalloc((void **)&d_coloring, num_points * sizeof(int)); 
      hipMalloc((void **)&d_reordering, num_points * sizeof(int)); 
      //hipMalloc((void **)&d_fraction, sizeof(float));

      //hipMemcpy(d_fraction, &fraction, sizeof(float), hipMemcpyHostToDevice); 
      //hipMemcpy(d_num_points,&num_points, sizeof(int), hipMemcpyHostToDevice);
      //hipMemcpy(d_nnz, &num_nonzero, sizeof(int), hipMemcpyHostToDevice);

      std::cout << "Mem alloc completed -- inside reorder2" << std::endl;
      hipsparseStatus_t status_color;

      hipsparseHandle_t handle_color;
      CusparseSafeCall(hipsparseCreate(&handle_color));

      hipsparseColorInfo_t info;
      status_color = hipsparseCreateColorInfo(&info);
      if (status_color != HIPSPARSE_STATUS_SUCCESS) {
        printf("error");
        exit(1);
      }
      START_IL_TIMER();
      status_color = hipsparseScsrcolor(handle_color, num_points, num_nonzero, sparse_matrix_descriptor, thrust::raw_pointer_cast(sparse_pij_device.data()), thrust::raw_pointer_cast(pij_row_ptr_device.data()), thrust::raw_pointer_cast(pij_col_ind_device.data()), &fraction, &ncolors, d_coloring, d_reordering, info );
      END_IL_TIMER(_time_perm);
      std::cout << "csrcolor completed " << std::endl;
       switch (status_color) {
          case HIPSPARSE_STATUS_SUCCESS:
            printf("success\n");
            break;
          case HIPSPARSE_STATUS_NOT_INITIALIZED:
            printf("not initialed\n");
          case HIPSPARSE_STATUS_ALLOC_FAILED:
            printf("alloc failed\n");
            break;
          case HIPSPARSE_STATUS_INVALID_VALUE:
            printf("invalid value\n");
            break;
          case HIPSPARSE_STATUS_ARCH_MISMATCH:
            printf("mismatch\n");
            break;
           case HIPSPARSE_STATUS_INTERNAL_ERROR:
            printf("internal error\n");
            break;
           case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            printf("not supported\n");
            break;
           default:
            printf("unknown error\n");
            break;
       }
       //hipMemcpy(&ncolors, d_ncolors, sizeof(int), hipMemcpyDeviceToHost);
       //printf("ncolors=%p, &ncolors=%p\n", d_ncolors, &d_ncolors);
       hipMemcpy(coloring, d_coloring, num_points * sizeof(int), hipMemcpyDeviceToHost);
       hipMemcpy(reordering, d_reordering, num_points * sizeof(int), hipMemcpyDeviceToHost);
      std::cout << "memcpy 1 finished color" << std::endl;
      std::ofstream reord_file;
      reord_file.open("reordering_color.txt");
      for (int i=0; i < num_points; i++) {
        reord_file << reordering[i] << " ";
      }

      int *h_mapBfromA = NULL;
      //float *h_pij_vals_b = NULL;
      //int *h_pij_col_ind_b = NULL;      
      hipsolverSpHandle_t sol_handle = NULL;
      checkCudaErrors(hipsolverSpCreate(&sol_handle));
       std::cout << "Created sparse solver handle" << std::endl;
      float *h_pij_vals = (float *)malloc((num_nonzero)*sizeof(float));
      //h_pij_vals = thrust::raw_pointer_cast(sparse_pij_device.data());
      hipMemcpy(h_pij_vals, thrust::raw_pointer_cast(sparse_pij_device.data()), sizeof(float)*(num_nonzero), hipMemcpyDeviceToHost);
      
      int *h_pij_row_ptr = (int *)malloc((num_points+1)*sizeof(int));
      //h_pij_row_ptr = thrust::raw_pointer_cast(pij_row_ptr_device.data());
      hipMemcpy(h_pij_row_ptr, thrust::raw_pointer_cast(pij_row_ptr_device.data()), sizeof(int)*(num_points+1), hipMemcpyDeviceToHost);

      int *h_pij_col_ind = (int *)malloc((num_nonzero)*sizeof(int));
      //h_pij_col_ind = thrust::raw_pointer_cast(pij_col_ind_device.data());
      hipMemcpy(h_pij_col_ind, thrust::raw_pointer_cast(pij_col_ind_device.data()), sizeof(int)*(num_nonzero), hipMemcpyDeviceToHost);

      //h_Q = (int *)malloc(sizeof(int)*num_points);
      //h_pij_row_ptr_b = (int *)malloc(sizeof(int)*(num_points+1));
      //h_pij_col_ind_b = (int *)malloc(sizeof(int)*(num_nonzero));
      //h_pij_vals_b = (float *)malloc(sizeof(float)*(num_nonzero));
      h_mapBfromA = (int *)malloc(sizeof(int)*num_nonzero);
      
      //check if memory has been allocated without any issues
      //assert(NULL != h_Q);
      //assert(NULL != h_pij_row_ptr_b);
      //assert(NULL != h_pij_col_ind_b);
      //assert(NULL != h_pij_vals_b   );
      assert(NULL != h_mapBfromA);
      
      std::cout << "Assertion done" << std::endl;

       size_t size_perm = 0;
       void *buffer_cpu = NULL;
       
       START_IL_TIMER();
       checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(sol_handle,num_points ,num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr,h_pij_col_ind, reordering, reordering, &size_perm));
       END_IL_TIMER(_time_reord_buff);

       buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
       assert(NULL!=buffer_cpu);
       for(int j = 0; j< num_nonzero; j++) {
        h_mapBfromA[j] = j;
       }

       START_IL_TIMER();
       checkCudaErrors(cusolverSpXcsrpermHost(sol_handle, num_points, num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, reordering, reordering, h_mapBfromA, buffer_cpu) );
       END_IL_TIMER(_time_reorder);
       START_IL_TIMER();

       std::vector<int> v_row_ptr(h_pij_row_ptr, h_pij_row_ptr + (num_points+1));
       if (h_pij_row_ptr) { free(h_pij_row_ptr);}
       thrust::host_vector<int> row_temp(v_row_ptr);

       std::vector<int> v_col_ind(h_pij_col_ind, h_pij_col_ind + (num_nonzero));
       if (h_pij_col_ind) {free(h_pij_col_ind);}
       thrust::host_vector<int> col_temp(v_col_ind);

       std::vector<float> v_vals(h_pij_vals, h_pij_vals + (num_nonzero+1));
       if(h_pij_vals) { free(h_pij_vals);}
       thrust::host_vector<float> vals_temp(v_vals);


       pij_row_ptr_device = row_temp;
       pij_col_ind_device = col_temp;
       sparse_pij_device = vals_temp;

       END_IL_TIMER(_time_devicecopy);

       std::cout << "Completed permuting" << std::endl;
    }
    else if(opt.reorder==8){
     int *h_mapBfromA = NULL;
      //float *h_pij_vals_b = NULL;
      //int *h_pij_col_ind_b = NULL;      
      hipsolverSpHandle_t sol_handle = NULL;
      checkCudaErrors(hipsolverSpCreate(&sol_handle));
       std::cout << "Created sparse solver handle" << std::endl;
      float *h_pij_vals = (float *)malloc((num_nonzero)*sizeof(float));
      //h_pij_vals = thrust::raw_pointer_cast(sparse_pij_device.data());
      hipMemcpy(h_pij_vals, thrust::raw_pointer_cast(sparse_pij_device.data()), sizeof(float)*(num_nonzero), hipMemcpyDeviceToHost);
      
      int *h_pij_row_ptr = (int *)malloc((num_points+1)*sizeof(int));
      //h_pij_row_ptr = thrust::raw_pointer_cast(pij_row_ptr_device.data());
      hipMemcpy(h_pij_row_ptr, thrust::raw_pointer_cast(pij_row_ptr_device.data()), sizeof(int)*(num_points+1), hipMemcpyDeviceToHost);

      int *h_pij_col_ind = (int *)malloc((num_nonzero)*sizeof(int));
      //h_pij_col_ind = thrust::raw_pointer_cast(pij_col_ind_device.data());
      hipMemcpy(h_pij_col_ind, thrust::raw_pointer_cast(pij_col_ind_device.data()), sizeof(int)*(num_nonzero), hipMemcpyDeviceToHost);

      //h_Q = (int *)malloc(sizeof(int)*num_points);
      //h_pij_row_ptr_b = (int *)malloc(sizeof(int)*(num_points+1));
      //h_pij_col_ind_b = (int *)malloc(sizeof(int)*(num_nonzero));
      //h_pij_vals_b = (float *)malloc(sizeof(float)*(num_nonzero));
      h_mapBfromA = (int *)malloc(sizeof(int)*num_nonzero);
      
      //check if memory has been allocated without any issues
      //assert(NULL != h_Q);
      //assert(NULL != h_pij_row_ptr_b);
      //assert(NULL != h_pij_col_ind_b);
      //assert(NULL != h_pij_vals_b   );
      assert(NULL != h_mapBfromA);

     int *h_Q = (int *)malloc(sizeof(int)*num_points);

     std::string line;
     std::ifstream myfile ("edg_perm.out");
     if(myfile.is_open())
     {
      int i = 0;
      while(std::getline(myfile,line)){
        h_Q[i] = std::atoi(line.c_str());
        i += 1;  
      }
       size_t size_perm = 0;
       void *buffer_cpu = NULL;
       
       START_IL_TIMER();
       checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(sol_handle,num_points ,num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr,h_pij_col_ind, h_Q, h_Q, &size_perm));
       END_IL_TIMER(_time_reord_buff);

       buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
       assert(NULL!=buffer_cpu);
       for(int j = 0; j< num_nonzero; j++) {
        h_mapBfromA[j] = j;
       }

       START_IL_TIMER();
       checkCudaErrors(cusolverSpXcsrpermHost(sol_handle, num_points, num_points, num_nonzero, sparse_matrix_descriptor, h_pij_row_ptr, h_pij_col_ind, h_Q, h_Q, h_mapBfromA, buffer_cpu) );
       END_IL_TIMER(_time_reorder);

     }
       std::vector<int> v_row_ptr(h_pij_row_ptr, h_pij_row_ptr + (num_points+1));
       if (h_pij_row_ptr) { free(h_pij_row_ptr);}
       thrust::host_vector<int> row_temp(v_row_ptr);

       std::vector<int> v_col_ind(h_pij_col_ind, h_pij_col_ind + (num_nonzero));
       if (h_pij_col_ind) {free(h_pij_col_ind);}
       thrust::host_vector<int> col_temp(v_col_ind);

       std::vector<float> v_vals(h_pij_vals, h_pij_vals + (num_nonzero+1));
       if(h_pij_vals) { free(h_pij_vals);}
       thrust::host_vector<float> vals_temp(v_vals);
       
       pij_row_ptr_device = row_temp;
       pij_col_ind_device = col_temp;
       sparse_pij_device = vals_temp;

    }
    else if(opt.reorder==9) {
      int *h_Q = (int *)malloc(sizeof(int)*num_points);

      std::string line;
      std::ifstream myfile ("edg_perm.out");
      if(myfile.is_open())
      {
        int i = 0;
        while(std::getline(myfile,line)){
          h_Q[i] = std::atoi(line.c_str());
          i += 1;  
        }

      tsnecuda::util::permuteCoo(gpu_opt, coo_indices_device, pij_row_ptr_device, pij_col_ind_device, h_Q, num_points, num_nonzero);
    }
    }
   END_IL_REORDER(_time_tot_perm);
    if(opt.reorder != 9){ 
      tsnecuda::util::Csr2Coo(gpu_opt, coo_indices_device, pij_row_ptr_device,
                            pij_col_ind_device, num_points, num_nonzero);
    }
    std::cout << "Num nonzero 2: " << num_nonzero << std::endl;
    
    tsnecuda::save_coo("coo_after_", coo_indices_device, num_nonzero);
    // FIT-TNSE Parameters
    int n_interpolation_points = 3;
    // float intervals_per_integer = 1;
    int min_num_intervals = 50;
    int N = num_points;
    // int D = 2;
    // The number of "charges" or s+2 sums i.e. number of kernel sums
    int n_terms = 4;
    int n_boxes_per_dim = min_num_intervals;

    // FFTW works faster on numbers that can be written as  2^a 3^b 5^c 7^d
    // 11^e 13^f, where e+f is either 0 or 1, and the other exponents are
    // arbitrary
    int allowed_n_boxes_per_dim[20] = {25,36, 50, 55, 60, 65, 70, 75, 80, 85, 90, 96, 100, 110, 120, 130, 140,150, 175, 200};
    if ( n_boxes_per_dim < allowed_n_boxes_per_dim[19] ) {
        //Round up to nearest grid point
        int chosen_i;
        for (chosen_i =0; allowed_n_boxes_per_dim[chosen_i]< n_boxes_per_dim; chosen_i++);
        n_boxes_per_dim = allowed_n_boxes_per_dim[chosen_i];
    }

    int n_total_boxes = n_boxes_per_dim * n_boxes_per_dim;
    int total_interpolation_points = n_total_boxes * n_interpolation_points * n_interpolation_points;
    int n_fft_coeffs_half = n_interpolation_points * n_boxes_per_dim;
    int n_fft_coeffs = 2 * n_interpolation_points * n_boxes_per_dim;
    int n_interpolation_points_1d = n_interpolation_points * n_boxes_per_dim;

    // FIT-TSNE Device Vectors
    thrust::device_vector<int> point_box_idx_device(N);
    thrust::device_vector<float> x_in_box_device(N);
    thrust::device_vector<float> y_in_box_device(N);
    thrust::device_vector<float> y_tilde_values(total_interpolation_points * n_terms);
    thrust::device_vector<float> x_interpolated_values_device(N * n_interpolation_points);
    thrust::device_vector<float> y_interpolated_values_device(N * n_interpolation_points);
    thrust::device_vector<float> potentialsQij_device(N * n_terms);
    thrust::device_vector<float> w_coefficients_device(total_interpolation_points * n_terms);
    thrust::device_vector<float> all_interpolated_values_device(
        n_terms * n_interpolation_points * n_interpolation_points * N);
    thrust::device_vector<float> output_values(
        n_terms * n_interpolation_points * n_interpolation_points * N);
    thrust::device_vector<int> all_interpolated_indices(
        n_terms * n_interpolation_points * n_interpolation_points * N);
    thrust::device_vector<int> output_indices(
        n_terms * n_interpolation_points * n_interpolation_points * N);
    thrust::device_vector<float> chargesQij_device(N * n_terms);
    thrust::device_vector<float> box_lower_bounds_device(2 * n_total_boxes);
    thrust::device_vector<float> box_upper_bounds_device(2 * n_total_boxes);
    thrust::device_vector<float> kernel_tilde_device(n_fft_coeffs * n_fft_coeffs);
    thrust::device_vector<thrust::complex<float>> fft_kernel_tilde_device(2 * n_interpolation_points_1d * 2 * n_interpolation_points_1d);
    thrust::device_vector<float> fft_input(n_terms * n_fft_coeffs * n_fft_coeffs);
    thrust::device_vector<thrust::complex<float>> fft_w_coefficients(n_terms * n_fft_coeffs * (n_fft_coeffs / 2 + 1));
    thrust::device_vector<float> fft_output(n_terms * n_fft_coeffs * n_fft_coeffs);

    // Easier to compute denominator on CPU, so we should just calculate y_tilde_spacing on CPU also
    float h = 1 / (float) n_interpolation_points;
    float y_tilde_spacings[n_interpolation_points];
    y_tilde_spacings[0] = h / 2;
    for (int i = 1; i < n_interpolation_points; i++) {
        y_tilde_spacings[i] = y_tilde_spacings[i - 1] + h;
    }
    float denominator[n_interpolation_points];
    for (int i = 0; i < n_interpolation_points; i++) {
        denominator[i] = 1;
        for (int j = 0; j < n_interpolation_points; j++) {
            if (i != j) {
                denominator[i] *= y_tilde_spacings[i] - y_tilde_spacings[j];
            }
        }
    }
    thrust::device_vector<float> y_tilde_spacings_device(y_tilde_spacings, y_tilde_spacings + n_interpolation_points);
    thrust::device_vector<float> denominator_device(denominator, denominator + n_interpolation_points);

    // Create the FFT Handles
    hipfftHandle plan_kernel_tilde, plan_dft, plan_idft;;
    CufftSafeCall(hipfftCreate(&plan_kernel_tilde));
    CufftSafeCall(hipfftCreate(&plan_dft));
    CufftSafeCall(hipfftCreate(&plan_idft));

    size_t work_size, work_size_dft, work_size_idft;
    int fft_dimensions[2] = {n_fft_coeffs, n_fft_coeffs};
    CufftSafeCall(hipfftMakePlan2d(plan_kernel_tilde, fft_dimensions[0], fft_dimensions[1], HIPFFT_R2C, &work_size));
    CufftSafeCall(hipfftMakePlanMany(plan_dft, 2, fft_dimensions,
                                    NULL, 1, n_fft_coeffs * n_fft_coeffs,
                                    NULL, 1, n_fft_coeffs * (n_fft_coeffs / 2 + 1),
                                    HIPFFT_R2C, n_terms, &work_size_dft));
    CufftSafeCall(hipfftMakePlanMany(plan_idft, 2, fft_dimensions,
                                    NULL, 1, n_fft_coeffs * (n_fft_coeffs / 2 + 1),
                                    NULL, 1, n_fft_coeffs * n_fft_coeffs,
                                    HIPFFT_C2R, n_terms, &work_size_idft));



    // Dump file
    float *host_ys = nullptr;
    std::ofstream dump_file;
    if (opt.get_dump_points()) {
        dump_file.open("pts_Y_" + std::to_string(opt.num_points));
        host_ys = new float[num_points * 2];
        dump_file << num_points << " " << 2 << std::endl;
    }

    #ifndef NO_ZMQ
        bool send_zmq = opt.get_use_interactive();
        zmq::context_t context(1);
        zmq::socket_t publisher(context, ZMQ_REQ);
        if (opt.get_use_interactive()) {

        // Try to connect to the socket
            if (opt.verbosity >= 1)
                std::cout << "Initializing Connection...." << std::endl;
            publisher.setsockopt(ZMQ_RCVTIMEO, opt.get_viz_timeout());
            publisher.setsockopt(ZMQ_SNDTIMEO, opt.get_viz_timeout());
            if (opt.verbosity >= 1)
                std::cout << "Waiting for connection to visualization for 10 secs...." << std::endl;
            publisher.connect(opt.get_viz_server());

            // Send the number of points we should be expecting to the server
            std::string message = std::to_string(opt.num_points);
            send_zmq = publisher.send(message.c_str(), message.length());

            // Wait for server reply
            zmq::message_t request;
            send_zmq = publisher.recv (&request);

            // If there's a time-out, don't bother.
            if (send_zmq) {
                if (opt.verbosity >= 1)
                    std::cout << "Visualization connected!" << std::endl;
            } else {
                std::cout << "No Visualization Terminal, continuing..." << std::endl;
                send_zmq = false;
            }
        }
    #else
        if (opt.get_use_interactive())
            std::cout << "This version is not built with ZMQ for interative viz. Rebuild with WITH_ZMQ=TRUE for viz." << std::endl;
    #endif

    if (opt.verbosity > 0) {
        std::cout << "done." << std::endl;
    }

    END_IL_TIMER(_time_init_fft);

    //create vector to record rep force computation time
    std::vector<float> rep_force_times;
    // Support for infinite iteration
    float time_mul, time_firstSPDM, time_secondSPDM, time_pijkern = 0.0;

    for (size_t step = 0; step != opt.iterations; step++) {

        START_IL_TIMER();
        float fill_value = 0;
        thrust::fill(w_coefficients_device.begin(), w_coefficients_device.end(), fill_value);
        thrust::fill(potentialsQij_device.begin(), potentialsQij_device.end(), fill_value);
        // Setup learning rate schedule
        if (step == opt.force_magnify_iters) {
            momentum = opt.post_exaggeration_momentum;
            attr_exaggeration = 1.0f;
        }
        END_IL_TIMER(_time_other);



        // Prepare the terms that we'll use to compute the sum i.e. the repulsive forces
        START_IL_TIMER();
        tsnecuda::ComputeChargesQij(chargesQij_device, points_device, num_points, n_terms);
        END_IL_TIMER(_time_compute_charges);

        // Compute Minimax elements
        START_IL_TIMER();
        auto minimax_iter = thrust::minmax_element(points_device.begin(), points_device.end());
        float min_coord = minimax_iter.first[0];
        float max_coord = minimax_iter.second[0];

        // Compute the number of boxes in a single dimension and the total number of boxes in 2d
        // auto n_boxes_per_dim = static_cast<int>(fmax(min_num_intervals, (max_coord - min_coord) / intervals_per_integer));

        tsnecuda::PrecomputeFFT2D(
            plan_kernel_tilde, max_coord, min_coord, max_coord, min_coord, n_boxes_per_dim, n_interpolation_points,
            box_lower_bounds_device, box_upper_bounds_device, kernel_tilde_device,
            fft_kernel_tilde_device);

        float box_width = ((max_coord - min_coord) / (float) n_boxes_per_dim);

        END_IL_TIMER(_time_precompute_2d);
        START_IL_TIMER();

        
        tsnecuda::NbodyFFT2D(
            plan_dft, plan_idft,
            N, n_terms, n_boxes_per_dim, n_interpolation_points,
            fft_kernel_tilde_device, n_total_boxes,
            total_interpolation_points, min_coord, box_width, n_fft_coeffs_half, n_fft_coeffs,
            fft_input, fft_w_coefficients, fft_output,
            point_box_idx_device, x_in_box_device, y_in_box_device, points_device,
            box_lower_bounds_device, y_tilde_spacings_device, denominator_device, y_tilde_values,
            all_interpolated_values_device, output_values, all_interpolated_indices,
            output_indices, w_coefficients_device, chargesQij_device, x_interpolated_values_device,
            y_interpolated_values_device, potentialsQij_device);

        END_IL_TIMER(_time_nbodyfft);
        rep_force_times.push_back(((float) duration.count()) / 1000000.0);
        START_IL_TIMER();

        // Make the negative term, or F_rep in the equation 3 of the paper
        normalization = tsnecuda::ComputeRepulsiveForces(
            repulsive_forces_device, normalization_vec_device, points_device,
            potentialsQij_device, num_points, n_terms);

        END_IL_TIMER(_time_norm);
        START_IL_TIMER();


        // Calculate Attractive Forces            
        tsnecuda::ComputeAttractiveForces(gpu_opt,
                                              sparse_handle,
                                              sparse_matrix_descriptor,
                                              attractive_forces_device,
					                                    //pijqij,
                                              sparse_pij_device,
                                              //d_sp_pij_re,
                                              pij_row_ptr_device,
                                              pij_col_ind_device,
                                              coo_indices_device,
                                              //d_coo_re,
                                              points_device,
                                              ones_device,
                                              num_points,
                                              time_firstSPDM,
                                              time_secondSPDM,
                                              time_mul,
                                              time_pijkern,
                                              num_nonzero);

        END_IL_TIMER(_time_attr);
        START_IL_TIMER();

        // Apply Forces
        tsnecuda::ApplyForces(gpu_opt,
                                  points_device,
                                  attractive_forces_device,
                                  repulsive_forces_device,
                                  gains_device,
                                  old_forces_device,
                                  eta,
                                  normalization,
                                  momentum,
                                  attr_exaggeration,
                                  num_points,
                                  num_blocks);
        END_IL_TIMER(_time_apply_forces);
        // // Compute the gradient norm
        tsnecuda::util::SquareDeviceVector(attractive_forces_device, old_forces_device);
        thrust::transform(attractive_forces_device.begin(), attractive_forces_device.begin()+num_points,
                          attractive_forces_device.begin()+num_points, attractive_forces_device.begin(),
                          thrust::plus<float>());
        tsnecuda::util::SqrtDeviceVector(attractive_forces_device, attractive_forces_device);
        float grad_norm = thrust::reduce(
            attractive_forces_device.begin(), attractive_forces_device.begin() + num_points,
            0.0f, thrust::plus<float>()) / num_points;
        thrust::fill(attractive_forces_device.begin(), attractive_forces_device.end(), 0.0f);
        //END_IL_TIMER(_time_apply_forces);

        if (grad_norm < opt.min_gradient_norm) {
            if (opt.verbosity >= 1) std::cout << "Reached minimum gradient norm: " << grad_norm << std::endl;
            break;
        }

        if (opt.verbosity >= 1 && step % opt.print_interval == 0) {
            std::cout << "[Step " << step << "] Avg. Gradient Norm: " << grad_norm << std::endl;
        }

        

        #ifndef NO_ZMQ
            if (send_zmq) {
            zmq::message_t message(sizeof(float)*opt.num_points*2);
            thrust::copy(points_device.begin(), points_device.end(), static_cast<float*>(message.data()));
            bool res = false;
            res = publisher.send(message);
            zmq::message_t request;
            res = publisher.recv(&request);
            if (!res) {
                std::cout << "Server Disconnected, Not sending anymore for this session." << std::endl;
            }
            send_zmq = res;
            }
        #endif

        if (opt.get_dump_points() && step % opt.get_dump_interval() == 0) {
            thrust::copy(points_device.begin(), points_device.end(), host_ys);
            for (int i = 0; i < opt.num_points; i++) {
                dump_file << host_ys[i] << " " << host_ys[i + num_points] << std::endl;
            }
        }

        // // Handle snapshoting
        if (opt.return_style == tsnecuda::RETURN_STYLE::SNAPSHOT && step % snap_interval == 0 && opt.return_data != nullptr) {
          thrust::copy(points_device.begin(),
                       points_device.end(),
                       snap_num*opt.num_points*2 + opt.return_data);
          snap_num += 1;
        }

    }

    CufftSafeCall(hipfftDestroy(plan_kernel_tilde));
    CufftSafeCall(hipfftDestroy(plan_dft));
    CufftSafeCall(hipfftDestroy(plan_idft));

    if (opt.verbosity > 0) {
        PRINT_IL_TIMER(_time_initialization);
        PRINT_IL_TIMER(_time_knn);
        PRINT_IL_TIMER(_time_knn2);
        PRINT_IL_TIMER(_time_normknn);
        PRINT_IL_TIMER(_time_symmetry);
        PRINT_IL_TIMER(_time_perm);
        PRINT_IL_TIMER(_time_reorder);
        PRINT_IL_TIMER(_time_reord_buff);
        PRINT_IL_TIMER(_time_mapping);
        PRINT_IL_TIMER(_time_hostcopy);
        PRINT_IL_TIMER(_time_devicecopy);
        PRINT_IL_TIMER(_time_tot_perm);
        PRINT_IL_TIMER(_time_init_low_dim);
        PRINT_IL_TIMER(_time_init_fft);
        PRINT_IL_TIMER(_time_compute_charges);
        PRINT_IL_TIMER(_time_precompute_2d);
        PRINT_IL_TIMER(_time_nbodyfft);
        PRINT_IL_TIMER(_time_norm);
        PRINT_IL_TIMER(_time_attr);
        PRINT_IL_TIMER(_time_apply_forces);
        PRINT_IL_TIMER(_time_other);
        PRINT_IL_TIMER(total_time);

        std::cout << "time_firstSPDM" << ": " << (time_firstSPDM) << "s" << std::endl;
        std::cout << "time_secondSPDM" << ": " << (time_secondSPDM) << "s" << std::endl;
        std::cout << "time_mul" << ": " << (time_mul) << "s" << std::endl;
        std::cout << "time_pijkern" << ": " << (time_pijkern ) << "s" << std::endl;


    }


    // Clean up the dump file if we are dumping points
    if (opt.get_dump_points()){
      delete[] host_ys;
      dump_file.close();
    }

    // Handle a once off return type
    if (opt.return_style == tsnecuda::RETURN_STYLE::ONCE && opt.return_data != nullptr) {
      thrust::copy(points_device.begin(), points_device.end(), opt.return_data);
    }

    // Handle snapshoting
    if (opt.return_style == tsnecuda::RETURN_STYLE::SNAPSHOT && opt.return_data != nullptr) {
      thrust::copy(points_device.begin(), points_device.end(), snap_num*opt.num_points*2 + opt.return_data);
    }
    if (opt.verbosity > 0) {
        std::ofstream reptimes_file;
        reptimes_file.open("./reptimes_" + std::to_string(opt.num_points/1000) + ".txt");
        //dump the values of sparse array Pij
        for (const auto &e : rep_force_times) reptimes_file << e << " ";
        //dump the indices of the values of Pij (COO format)
        //for (const auto &e : stl_pij_coo) pij_file << e << " ";
        //dump reordered values of sparse array pij
        //for(const auto &e : stl_reordered_pij) pij_file << e << " ";
        //dump the reordered indices of the values of Pij
        //for(const auto &e : stl_reordered_coo) pij_file << e << " ";

        reptimes_file.close();

    }
   
    // Return some final values
    opt.trained = true;
    opt.trained_norm = normalization;

    return;
}
