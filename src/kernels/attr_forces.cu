#include "hip/hip_runtime.h"
// TODO: add copyright

/*
    Compute unnormalized attractive force for barnes-hut approximation of t-SNE.

    Attractive force is given by pij*qij.
*/

#include "kernels/attr_forces.h"

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed with error (%d) at line %d\n",             \
               status, __LINE__);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}


__global__
void ComputePijxQijKernel(
                            float * __restrict__ attr_forces,
                            const float * __restrict__ pij,
                            const float * __restrict__ points,
                            const int * __restrict__ coo_indices,
                            const int num_points,
                            const int num_nonzero)
{
    register int TID, i, j;
    register float ix, iy, jx, jy, dx, dy, pijqij;
    TID = threadIdx.x + blockIdx.x * blockDim.x;
    if (TID >= num_nonzero) return;
    i = coo_indices[2*TID];
    j = coo_indices[2*TID+1];

    ix = points[i]; iy = points[num_points + i];
    jx = points[j]; jy = points[num_points + j];
    dx = ix - jx;
    dy = iy - jy;
    pijqij = pij[TID] / (1 + dx*dx + dy*dy);
    atomicAdd(attr_forces + i, pijqij * dx);
    atomicAdd(attr_forces + num_points + i, pijqij * dy);
}

void tsnecuda::ComputeAttractiveForces(
                    tsnecuda::GpuOptions &gpu_opt,
                    thrust::device_vector<float> &attr_forces,
                    //thrust::device_vector<float> &pijqij,
                    thrust::device_vector<float> &sparse_pij,
                    thrust::device_vector<int> &pij_row_ptr,
                    thrust::device_vector<int> &pij_col_ind,
                    thrust::device_vector<int> &coo_indices,
                    thrust::device_vector<float> &points,
                    const int num_points,
                    const int num_nonzero)
{
    // Computes pij*qij for each i,j
    // TODO: this is bad style
    //
    
    //init timers
    //auto time_pijkern_ = duration;
    //auto time_firstSPDM_ = duration;
    //auto time_secondSPDM_ = duration;
    //auto time_mul_ = duration;

    const int BLOCKSIZE = 1024;
    const int NBLOCKS = iDivUp(num_nonzero, BLOCKSIZE);
    
    //START_IL_TIMER();

    ComputePijxQijKernel<<<NBLOCKS, BLOCKSIZE>>>(
                    thrust::raw_pointer_cast(attr_forces.data()),
                    thrust::raw_pointer_cast(sparse_pij.data()),
                    thrust::raw_pointer_cast(points.data()),
                    thrust::raw_pointer_cast(coo_indices.data()),
                    num_points,
                    num_nonzero);
    GpuErrorCheck(hipDeviceSynchronize());
    
    //END_IL_TIMER(time_pijkern_);
    //size_t bufferSize = 0;
    //void* dBuffer = NULL;
    
    //float alpha = 1.0f;
    //float beta = 0.0f;
    
    //START_IL_TIMER();
    // (PijxQij)*(Ones)
    //hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
          //num_points, 2, num_points, num_nonzero, &alpha, descrSp,
          //thrust::raw_pointer_cast(pijqij.data()),
          //thrust::raw_pointer_cast(pij_row_ptr.data()),
          //thrust::raw_pointer_cast(pij_col_ind.data()),
        //  thrust::raw_pointer_cast(ones.data()), num_points, &beta,
      //    thrust::raw_pointer_cast(attr_forces.data()), num_points);

    //GpuErrorCheck(hipDeviceSynchronize());
    //END_IL_TIMER(time_firstSPDM_);
    // The first Hadamard product
    //START_IL_TIMER(); 
    //thrust::transform(attr_forces.begin(), attr_forces.end(), points.begin(),
     //   attr_forces.begin(), thrust::multiplies<float>());

    //END_IL_TIMER(time_mul_);
    //alpha = -1.0f;
    //beta = 1.0f;
    
    //START_IL_TIMER();
    // (PijxQij)*Y
    //hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //      num_points, 2, num_points, num_nonzero, &alpha, descrSp,
    //      thrust::raw_pointer_cast(pijqij.data()),
    //      thrust::raw_pointer_cast(pij_row_ptr.data()),
    //      thrust::raw_pointer_cast(pij_col_ind.data()),
    //      thrust::raw_pointer_cast(points.data()), num_points, &beta,
    //      thrust::raw_pointer_cast(attr_forces.data()), num_points);

    //END_IL_TIMER(time_secondSPDM_);

    //time_firstSPDM = ((float) time_firstSPDM_.count()) / 1000000.0;
    //time_secondSPDM = ((float) time_secondSPDM_.count()) / 1000000.0;  
    //time_mul = ((float) time_mul_.count()) / 1000000.0; 
    //time_pijkern = ((float) time_pijkern_.count()) / 1000000.0; 
}
