#include "hip/hip_runtime.h"
// TODO: add copyright

/*
    Compute unnormalized attractive force for barnes-hut approximation of t-SNE.

    Attractive force is given by pij*qij.
*/

#include "kernels/attr_forces.h"
#include <chrono>
#define START_IL_TIMER() start = std::chrono::high_resolution_clock::now();
#define END_IL_TIMER(x) stop = std::chrono::high_resolution_clock::now(); duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start); x += duration;

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed with error (%d) at line %d\n",             \
               status, __LINE__);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}


__global__
void ComputePijxQijKernel(
                            volatile float * __restrict__ pijqij,
                            const float * __restrict__ pij,
                            const float * __restrict__ points,
                            const int * __restrict__ coo_indices,
                            const int num_points,
                            const int num_nonzero)
{
    register int TID, i, j;
    register float ix, iy, jx, jy, dx, dy;
    TID = threadIdx.x + blockIdx.x * blockDim.x;
    if (TID >= num_nonzero) return;
    i = coo_indices[2*TID];
    j = coo_indices[2*TID+1];

    ix = points[i]; iy = points[num_points + i];
    jx = points[j]; jy = points[num_points + j];
    dx = ix - jx;
    dy = iy - jy;
    pijqij[TID] = pij[TID] / (1 + dx*dx + dy*dy);
    //atomicAdd(attr_forces + i, pijqij * dx);
    //atomicAdd(attr_forces + num_points + i, pijqij * dy);
}
void tsnecuda::ComputeAttractiveForcesBSR(
                    tsnecuda::GpuOptions &gpu_opt,
                    hipsparseHandle_t &handle,
                    hipsparseMatDescr_t &bsr_descr,
                    thrust::device_vector<float> &attr_forces,
                    thrust::device_vector<float> &pijqij,
                    thrust::device_vector<float> &sparse_pij_device,
                    float *bsrVal,
                    int *bsrRowPtr,
                    int *bsrColInd,
                    thrust::device_vector<int> &coo_indices,
                    thrust::device_vector<float> &points,
                    thrust::device_vector<float> &ones,
                    const int num_points,
                    const int num_nonzero,
                    const int nnzb)
{
   const int BLOCKSIZE = 1024;
    const int NBLOCKS = iDivUp(num_nonzero, BLOCKSIZE);
    
    //START_IL_TIMER();

    ComputePijxQijKernel<<<NBLOCKS, BLOCKSIZE>>>(
                    thrust::raw_pointer_cast(pijqij.data()),
                    thrust::raw_pointer_cast(sparse_pij_device.data()),
                    thrust::raw_pointer_cast(points.data()),
                    thrust::raw_pointer_cast(coo_indices.data()),
                    num_points,
                    num_nonzero);
    GpuErrorCheck(hipDeviceSynchronize());
    
    float alpha = 1.0f;
    float beta = 0.0f;
    
    int mb = (num_points + BLOCKSIZE-1)/BLOCKSIZE;

    const int m = mb * BLOCKSIZE;
    const int ldb = m;
    const int ldc = m;

    hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb,
        2, mb, nnzb, &alpha, bsr_descr, bsrVal, bsrRowPtr, bsrColInd,
        BLOCKSIZE, thrust::raw_pointer_cast(ones.data()), ldb, &beta,
        thrust::raw_pointer_cast(attr_forces.data()), ldc );
    GpuErrorCheck(hipDeviceSynchronize());

    //Second Hadamard Prod.
    thrust::transform(attr_forces.begin(), attr_forces.end(), points.begin(),
        attr_forces.begin(), thrust::multiplies<float>());
    GpuErrorCheck(hipDeviceSynchronize());

    hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb,
        2, mb, nnzb, &alpha, bsr_descr, bsrVal, bsrRowPtr, bsrColInd,
        BLOCKSIZE, thrust::raw_pointer_cast(points.data()), ldb, &beta,
        thrust::raw_pointer_cast(attr_forces.data()), ldc );
    GpuErrorCheck(hipDeviceSynchronize());

}

void tsnecuda::ComputeAttractiveForces(
                    tsnecuda::GpuOptions &gpu_opt,
                    hipsparseHandle_t &handle,
                    hipsparseMatDescr_t &descrSp,
                    thrust::device_vector<float> &attr_forces,
                    thrust::device_vector<float> &pijqij,
                    thrust::device_vector<float> &sparse_pij,
                    thrust::device_vector<int> &pij_row_ptr,
                    thrust::device_vector<int> &pij_col_ind,
                    thrust::device_vector<int> &coo_indices,
                    thrust::device_vector<float> &points,
                    thrust::device_vector<float> &ones,
                    const int num_points,
                    float &time_firstSPDM,
                    float &time_secondSPDM,
                    float &time_mul,
                    float &time_pijkern,
                    const int num_nonzero)
{
    // Computes pij*qij for each i,j
    // TODO: this is bad style
    //
    
    auto start = std::chrono::high_resolution_clock::now();
    auto stop = std::chrono::high_resolution_clock::now();
    
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    
    //init timers
    auto time_pijkern_ = duration;
    auto time_firstSPDM_ = duration;
    auto time_secondSPDM_ = duration;
    auto time_mul_ = duration;

    const int BLOCKSIZE = 1024;
    const int NBLOCKS = iDivUp(num_nonzero, BLOCKSIZE);
    
    START_IL_TIMER();

    ComputePijxQijKernel<<<NBLOCKS, BLOCKSIZE>>>(
                    thrust::raw_pointer_cast(pijqij.data()),
                    thrust::raw_pointer_cast(sparse_pij.data()),
                    thrust::raw_pointer_cast(points.data()),
                    thrust::raw_pointer_cast(coo_indices.data()),
                    num_points,
                    num_nonzero);
    GpuErrorCheck(hipDeviceSynchronize());
    
    END_IL_TIMER(time_pijkern_);
    //size_t bufferSize = 0;
    //void* dBuffer = NULL;
    
    float alpha = 1.0f;
    float beta = 0.0f;
    
    START_IL_TIMER();
    // (PijxQij)*(Ones)
    hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
          num_points, 2, num_points, num_nonzero, &alpha, descrSp,
          thrust::raw_pointer_cast(pijqij.data()),
          thrust::raw_pointer_cast(pij_row_ptr.data()),
          thrust::raw_pointer_cast(pij_col_ind.data()),
          thrust::raw_pointer_cast(ones.data()), num_points, &beta,
          thrust::raw_pointer_cast(attr_forces.data()), num_points);

    GpuErrorCheck(hipDeviceSynchronize());
    END_IL_TIMER(time_firstSPDM_);
    // The first Hadamard product
    START_IL_TIMER(); 
    thrust::transform(attr_forces.begin(), attr_forces.end(), points.begin(),
        attr_forces.begin(), thrust::multiplies<float>());

    END_IL_TIMER(time_mul_);
    alpha = -1.0f;
    beta = 1.0f;
    
    START_IL_TIMER();
    // (PijxQij)*Y
    hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
          num_points, 2, num_points, num_nonzero, &alpha, descrSp,
          thrust::raw_pointer_cast(pijqij.data()),
          thrust::raw_pointer_cast(pij_row_ptr.data()),
          thrust::raw_pointer_cast(pij_col_ind.data()),
          thrust::raw_pointer_cast(points.data()), num_points, &beta,
          thrust::raw_pointer_cast(attr_forces.data()), num_points);

    END_IL_TIMER(time_secondSPDM_);

    time_firstSPDM = ((float) time_firstSPDM_.count()) / 1000000.0;
    time_secondSPDM = ((float) time_secondSPDM_.count()) / 1000000.0;  
    time_mul = ((float) time_mul_.count()) / 1000000.0; 
    time_pijkern = ((float) time_pijkern_.count()) / 1000000.0; 
}
